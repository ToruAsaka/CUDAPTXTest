#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

union  LoadData {
	long long Load;
	short DivData[4];
};

__device__ short DeviceMem[10000];
__global__ void LoadCheck_Rem(int Ret) {
	int x = threadIdx.x;
	unsigned long long LoadReg;
	unsigned long long RefReg = 0x000000000000FFFF;
	long long * lpDev = (long long  *)DeviceMem;
	LoadReg = __ldg(&lpDev[x]);
	int Clock;
	asm volatile (
		"mov.u32 %0, %clock;\n\t"
		:"=r"(Clock)
		);

	__shared__ short Cost[4];
	Cost[0] = (short)((LoadReg & RefReg)	  );
	RefReg <<= 16;
 	Cost[1] = (short)((LoadReg &  RefReg)>>16);
	RefReg <<= 16;
	Cost[2] = (short)((LoadReg & RefReg) >> 32);
	RefReg <<= 16;
	Cost[3] = (short)((LoadReg & RefReg) >> 48);
	int minCost = Cost[0];
	int Ref;
	int RefFwdData = 1;
	for (int nLoop = 1; nLoop < 1000; nLoop++) {
		int Ref = nLoop % 4;
		if (Ref == 1) {
			LoadReg = lpDev[x + RefFwdData++];
		}
		if (Cost[0] < minCost) {
			minCost = Cost[0];
		}
		if (x + 1 + nLoop >= 10000) {
			break;
		}
		Cost[0] = Cost[1];
		Cost[1] = Cost[2];
		Cost[2] = Cost[3];
		Cost[3] = 0;
		if (Ref == 0) {
			// Get short on Reg 
			Cost[0] = (short)((LoadReg & 0x000000000000FFFF)		);
			Cost[1] = (short)((LoadReg & 0x00000000FFFF0000) >> 16	);
			Cost[2] = (short)((LoadReg & 0x0000FFFF00000000) >> 32	);
			Cost[3] = (short)((LoadReg & 0xFFFF000000000000) >> 48	);
		}
	}
}
__global__ void LoadCheck(int Ret) {
	int x = threadIdx.x;
	short Cost = DeviceMem[x];
	int minCost = Cost;
	int Ref;
	int RefFwdData = 1;
	if (x + 1  < 10000) {
		for (int nLoop = 1; nLoop < 1000; nLoop++) {
			Cost = DeviceMem[x+nLoop];
			if (Cost < minCost) {
				minCost = Cost;
			}
		}
	}
}
int main()
{
	int x, y, z;
	x = 0;
	y = 0;
	z = 0;
	LoadCheck << <1, 1024 >> > (x);
	LoadCheck_Rem << <1, 1024 >> > (x);

    return 0;
}
